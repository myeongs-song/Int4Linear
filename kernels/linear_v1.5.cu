#ifndef _CUSTOM_INT4_LINEAR_CU
#define _CUSTOM_INT4_LINEAR_CU

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "linear_v1.5.h"

void linear_v1_5_launch(
    void *x_packed_d, void *w_packed_d, void *y_d, int m, int n, int k
) {
    using config = LinearConfig<7, 6, 7, 5, 4>;

    constexpr int n_threads = config::kWarpsPerThreadblockM * config::kWarpsPerThreadblockN * (1 << LOG2_WARP_SIZE);
    constexpr int out_tile_size_m = config::kThreadblockShapeM;
    constexpr int out_tile_size_n = config::kThreadblockShapeN;
    int n_blocks_m = (m+out_tile_size_m-1)/out_tile_size_m;
    int n_blocks_n = (n+out_tile_size_n-1)/out_tile_size_n;
    dim3 blockDim(n_threads, 1);
    dim3 gridDim(n_blocks_n, n_blocks_m);

    LinearArgs args = {
        .X_ptr = x_packed_d,
        .W_ptr = w_packed_d,
        .Y_ptr = y_d,
        .m = m, .n = n, .k = k
    };

    linear_v1_5_kernel<config><<<gridDim, blockDim>>>(args);
    
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA FAILED: %s\n", hipGetErrorString(err));
    }
    else printf("CUDA SUCCESS!\n");

}


#endif